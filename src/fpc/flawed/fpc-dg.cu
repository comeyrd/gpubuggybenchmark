#include "hip/hip_runtime.h"
#include <stdio.h>      /* defines printf for tests */
#include <stdlib.h> 
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpc-dg.hpp"
#include "cuda-utils.hpp"
//Data-locality - Using global for shared memory
//TODO

namespace {
__device__
unsigned my_abs ( int x )
{
  unsigned t = x >> 31;
  return (x ^ t) - t;
}
}

__global__ void
fpc_dg_kernel(const ulong *values, unsigned *cmp_size, size_t* length,unsigned* compressable) {
    int lid = threadIdx.x;
    int WGS = blockDim.x;
    int block_ix = blockIdx.x;
    size_t gid = block_ix * WGS + lid;
    if(gid > *length) return;
    ulong value = values[gid];
    unsigned inc;

    // 000
    if (value == 0) {
        inc = 1;
    }
    // 001 010
    else if ((my_abs((int)(value)) <= 0xFF)) {
        inc = 1;
    }
    // 011
    else if ((my_abs((int)(value)) <= 0xFFFF)) {
        inc = 2;
    }
    // 100
    else if ((((value) & 0xFFFF) == 0)) {
        inc = 2;
    }
    // 101
    else if ((my_abs((int)((value) & 0xFFFF))) <= 0xFF && my_abs((int)((value >> 16) & 0xFFFF)) <= 0xFF) {
        inc = 2;
    }
    // 110
    else if ((((value) & 0xFF) == ((value >> 8) & 0xFF)) &&
             (((value) & 0xFF) == ((value >> 16) & 0xFF)) &&
             (((value) & 0xFF) == ((value >> 24) & 0xFF))) {
        inc = 1;
    } else {
        inc = 4;
    }

    if (lid == 0)
        compressable[block_ix] = 0;
    __syncthreads();

    atomicAdd(&compressable[block_ix], inc);
    __syncthreads();
    if (lid == WGS - 1) {
        atomicAdd(cmp_size, compressable[block_ix]);
    }
}

KernelStats DGFPC::run(const FPCData &data, const FPCSettings &settings, FPCResult &result) const {
  CudaProfiling prof;
  ulong* d_values;
  unsigned* d_cmp_size;
  unsigned* d_compressable;
  size_t* d_length;
  dim3 grids (data.length/settings.wgz);
  dim3 threads (settings.wgz);
  prof.begin_mem2D();
  CHECK_CUDA(hipMalloc((void**)&d_values, data.b_size));
  CHECK_CUDA(hipMemcpy(d_values, data.values, data.b_size, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMalloc((void**)&d_cmp_size, sizeof(unsigned)));
  CHECK_CUDA(hipMalloc((void**)&d_length, sizeof(size_t)));
  CHECK_CUDA(hipMemcpy(d_length, &data.length, sizeof(size_t), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMalloc((void**)&d_compressable, sizeof(unsigned) * grids.x));

  prof.end_mem2D();

  prof.begin_compute();
  for(int r = 0 ; r < settings.repetitions ; r++){
    CHECK_CUDA(hipMemset(d_cmp_size, 0, sizeof(int)));
    fpc_dg_kernel<<<grids, threads>>>(d_values, d_cmp_size,d_length,d_compressable);
    hipError_t err = hipGetLastError();  // check launch errors
    if (err != hipSuccess) {
        printf("CUDA launch error: %s\n", hipGetErrorString(err));
    }
  }
  prof.end_compute();
  prof.begin_mem2H();
  CHECK_CUDA(hipMemcpy(&result.size_, d_cmp_size, sizeof(unsigned), hipMemcpyDeviceToHost));
  prof.end_mem2H();
  CHECK_CUDA(hipFree(d_values));
  CHECK_CUDA(hipFree(d_length));
  CHECK_CUDA(hipFree(d_cmp_size));
  return prof.retreive(settings.repetitions);
};

REGISTER_CLASS(IFPC,DGFPC)