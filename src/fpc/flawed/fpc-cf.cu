#include "hip/hip_runtime.h"
#include <stdio.h>      /* defines printf for tests */
#include <stdlib.h> 
#include <chrono>
#include <hip/hip_runtime.h>
#include "fpc-cf.hpp"
#include "cuda-utils.hpp"
//Inneficient Cache access -  False sharing (sharing cache line)

namespace {
__device__
unsigned my_abs ( int x )
{
  unsigned t = x >> 31;
  return (x ^ t) - t;
}
}

__global__ void
fpc_cf_kernel(const ulong *values, unsigned *cmp_size, size_t* length) {
    __shared__ unsigned compressable[100];    
    int lid = threadIdx.x;
    int WGS = blockDim.x;
    size_t gid = blockIdx.x * WGS + lid;
    if(gid > *length) return;
    ulong value = values[gid];
    unsigned inc;

    // 000
    if (value == 0) {
        inc = 1;
    }
    // 001 010
    else if ((my_abs((int)(value)) <= 0xFF)) {
        inc = 1;
    }
    // 011
    else if ((my_abs((int)(value)) <= 0xFFFF)) {
        inc = 2;
    }
    // 100
    else if ((((value) & 0xFFFF) == 0)) {
        inc = 2;
    }
    // 101
    else if ((my_abs((int)((value) & 0xFFFF))) <= 0xFF && my_abs((int)((value >> 16) & 0xFFFF)) <= 0xFF) {
        inc = 2;
    }
    // 110
    else if ((((value) & 0xFF) == ((value >> 8) & 0xFF)) &&
             (((value) & 0xFF) == ((value >> 16) & 0xFF)) &&
             (((value) & 0xFF) == ((value >> 24) & 0xFF))) {
        inc = 1;
    } else {
        inc = 4;
    }

    if (lid < 100)
        compressable[lid] = 0;
    __syncthreads();
    atomicAdd(&compressable[lid%100], inc);
    __syncthreads();
    if (lid < 100) {
        atomicAdd(cmp_size, compressable[lid]);
    }
}

KernelStats CFFPC::run(const FPCData &data, const FPCSettings &settings, FPCResult &result) const {
  CudaProfiling prof(settings);
  ulong* d_values;
  unsigned* d_cmp_size;
  size_t* d_length;
  prof.begin_mem2D();
  CHECK_CUDA(hipMalloc((void**)&d_values, data.b_size));
  CHECK_CUDA(hipMemcpy(d_values, data.values, data.b_size, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMalloc((void**)&d_cmp_size, sizeof(unsigned)));
  CHECK_CUDA(hipMalloc((void**)&d_length, sizeof(size_t)));
  CHECK_CUDA(hipMemcpy(d_length, &data.length, sizeof(size_t), hipMemcpyHostToDevice));

  prof.end_mem2D();
  dim3 grids (data.length/settings.wgz);
  dim3 threads (settings.wgz);
  for(int w = 0; w < settings.warmup ; w++){
    prof.begin_warmup();
    CHECK_CUDA(hipMemset(d_cmp_size, 0, sizeof(int)));
    fpc_cf_kernel<<<grids, threads>>>(d_values, d_cmp_size,d_length);
    prof.end_warmup();
  }
  for(int r = 0 ; r < settings.repetitions ; r++){
    prof.begin_repetition();
    CHECK_CUDA(hipMemset(d_cmp_size, 0, sizeof(int)));
    fpc_cf_kernel<<<grids, threads>>>(d_values, d_cmp_size,d_length);
    prof.end_repetition();
  }
  prof.begin_mem2H();
  CHECK_CUDA(hipMemcpy(&result.size_, d_cmp_size, sizeof(unsigned), hipMemcpyDeviceToHost));
  prof.end_mem2H();
  CHECK_CUDA(hipFree(d_values));
  CHECK_CUDA(hipFree(d_length));
  CHECK_CUDA(hipFree(d_cmp_size));
  return prof.retreive();
};

REGISTER_CLASS(IFPC,CFFPC)