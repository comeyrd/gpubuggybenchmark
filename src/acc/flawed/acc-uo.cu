#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "acc-uo.hpp"
#include "cuda-utils.hpp"
//Unnecessary Operation, computation that is not (anymore) used in the algorithm

#define GPU_NUM_THREADS 256

template <typename T>
__device__ void BlockReduce(T &input) {
  typedef hipcub::BlockReduce<T, GPU_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  input = BlockReduce(temp_storage).Sum(input);
}

__global__
void accuracy_uo_kernel(const int N, const int D, const int top_k, const float* Xdata, const int* labelData, int* accuracy){
  int count = 0;
  int n_count = 0;
  for (int row = blockIdx.x; row < N; row += gridDim.x) {
    const int label = labelData[row];
    const float label_pred = Xdata[row * D + label];
    int ngt = 0;
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      const float pred = Xdata[row * D + col];
      if (pred > label_pred || (pred == label_pred && col <= label)) {
        ++ngt;
      }else{
        ++n_count;//BUG
      }
    }
    BlockReduce(ngt);
    if (ngt <= top_k) {
      ++count;
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) { 
    atomicAdd(accuracy, count);
  }
}

KernelStats UOAccuracy::accuracy(const AccuracyData &aData, const AccuracySettings &aSettings, AccuracyResult &aResult) const{
    CudaProfiling prof;

    prof.begin_mem2D();
    int *d_label;
    CHECK_CUDA(hipMalloc((void**)&d_label, aData.label_sz_bytes));
    CHECK_CUDA(hipMemcpy(d_label, aData.label, aData.label_sz_bytes, hipMemcpyHostToDevice));

    float *d_data;
    CHECK_CUDA(hipMalloc((void**)&d_data, aData.label_sz_bytes));
    CHECK_CUDA(hipMemcpy(d_data, aData.data, aData.label_sz_bytes, hipMemcpyHostToDevice));

    int *d_count;
    CHECK_CUDA(hipMalloc((void**)&d_count, sizeof(int)));

    dim3 block (GPU_NUM_THREADS);

    dim3 grid (aSettings.grid_sz);

    CHECK_CUDA(hipMemset(d_count, 0, sizeof(int)));
    prof.end_mem2D();
    prof.begin_compute();
    accuracy_uo_kernel<<<grid, block>>>(aData.n_rows, aData.ndims, aData.topk, d_data, d_label, d_count);
    prof.end_compute();

    CHECK_CUDA(hipDeviceSynchronize());
    prof.begin_mem2H();
    CHECK_CUDA(hipMemcpy(&aResult.count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    prof.end_mem2H();

    CHECK_CUDA(hipFree(d_label));
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_count));
    return prof.retreive();
};

REGISTER_CLASS(IAccuracy,UOAccuracy);