#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "acc-dr.hpp"
#include "cuda-utils.hpp"
//Data Locality, using register memory for shared or global memory (register spilling)
//TODO FIX
#define GPU_NUM_THREADS 256

template <typename T>
__device__ void BlockReduce(T &input) {
  typedef hipcub::BlockReduce<T, GPU_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  input = BlockReduce(temp_storage).Sum(input);
}

__global__
void accuracy_dr_kernel(const int N, const int D, const int top_k, const float* Xdata, const int* labelData, int* accuracy){
  int count = 0;
  float l_xdata[10000] ;
  for (int i = threadIdx.x; i < D*N; i += blockDim.x) {
    l_xdata[i] = Xdata[i];
  }

  __syncthreads();
  for(int row = blockIdx.x; row < N; row += gridDim.x) {
    const int label = labelData[row];
    const float label_pred = l_xdata[row * D + label];
    int ngt = 0;
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      const float pred = l_xdata[row * D + col];
      if (pred > label_pred || (pred == label_pred && col <= label)) {
        ++ngt;
      }
    }
    BlockReduce(ngt);
    if (ngt <= top_k) {
      ++count;
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) { 
    atomicAdd(accuracy, count);
  }
}

KernelStats DRAccuracy::run(const AccuracyData &data, const AccuracySettings &settings, AccuracyResult &result) const{
    CudaProfiling prof(settings);

    prof.begin_mem2D();
    int *d_label;
    CHECK_CUDA(hipMalloc((void**)&d_label, data.label_sz_bytes));
    CHECK_CUDA(hipMemcpy(d_label, data.label, data.label_sz_bytes, hipMemcpyHostToDevice));
    
    float *d_data;
    CHECK_CUDA(hipMalloc((void**)&d_data, data.data_sz_bytes));
    CHECK_CUDA(hipMemcpy(d_data, data.data, data.label_sz_bytes, hipMemcpyHostToDevice));

    int *d_count;
    CHECK_CUDA(hipMalloc((void**)&d_count, sizeof(int)));

    dim3 block (GPU_NUM_THREADS);

    dim3 grid (settings.grid_sz);

    
    prof.end_mem2D();
    for(int w = 0; w < settings.warmup ; w++){
      prof.begin_warmup();
      CHECK_CUDA(hipMemset(d_count, 0, sizeof(int)));
      accuracy_dr_kernel<<<grid, block>>>(data.n_rows, data.ndims, data.topk, d_data, d_label, d_count);
      prof.end_warmup();
    }
    for(int r = 0 ; r < settings.repetitions ; r++){
      prof.begin_repetition();
      CHECK_CUDA(hipMemset(d_count, 0, sizeof(int)));
      accuracy_dr_kernel<<<grid, block>>>(data.n_rows, data.ndims, data.topk, d_data, d_label, d_count);
      prof.end_repetition();
    }

    CHECK_CUDA(hipDeviceSynchronize());
    prof.begin_mem2H();
    CHECK_CUDA(hipMemcpy(&result.count, d_count, sizeof(int), hipMemcpyDeviceToHost));
    prof.end_mem2H();

    CHECK_CUDA(hipFree(d_label));
    CHECK_CUDA(hipFree(d_data));
    CHECK_CUDA(hipFree(d_count));
    return prof.retreive();
};

//REGISTER_CLASS(IAccuracy,DRAccuracy);