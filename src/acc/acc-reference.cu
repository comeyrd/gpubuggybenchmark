#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "acc-reference.hpp"
#include "cuda-utils.hpp"

#define GPU_NUM_THREADS 256
//TODO Move the template inside the accuracy kernel and enum with possible gpu num threads

template <typename T>
__device__ void BlockReduce(T &input) {
  typedef hipcub::BlockReduce<T, GPU_NUM_THREADS> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  input = BlockReduce(temp_storage).Sum(input);
}

__global__
void accuracy_reference_kernel(const int N, const int D, const int top_k, const float* Xdata, const int* labelData, int* accuracy){
  int count = 0;

  for (int row = blockIdx.x; row < N; row += gridDim.x) {
    const int label = labelData[row];
    const float label_pred = Xdata[row * D + label];
    int ngt = 0;
    for (int col = threadIdx.x; col < D; col += blockDim.x) {
      const float pred = Xdata[row * D + col];
      if (pred > label_pred || (pred == label_pred && col <= label)) {
        ++ngt;
      }
    }
    BlockReduce(ngt);
    if (ngt <= top_k) {
      ++count;
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) { 
    atomicAdd(accuracy, count);
  }
}

KernelStats ReferenceAccuracy::accuracy(const AccuracyData &aData, const AccuracySettings &aSettings, AccuracyResult &aResult) const{
    CudaProfiling prof;

    prof.begin_mem2D();
    int *d_label;
    hipMalloc((void**)&d_label, aData.label_sz_bytes);
    hipMemcpy(d_label, aData.label, aData.label_sz_bytes, hipMemcpyHostToDevice);

    float *d_data;
    hipMalloc((void**)&d_data, aData.label_sz_bytes);
    hipMemcpy(d_data, aData.data, aData.label_sz_bytes, hipMemcpyHostToDevice);

    int *d_count;
    hipMalloc((void**)&d_count, sizeof(int));

    dim3 block (GPU_NUM_THREADS);

    dim3 grid (aSettings.grid_sz);

    hipMemset(d_count, 0, sizeof(int));
    prof.end_mem2D();
    prof.begin_compute();
    accuracy_reference_kernel<<<grid, block>>>(aData.n_rows, aData.ndims, aData.topk, d_data, d_label, d_count);
    prof.end_compute();

    CHECK_CUDA(hipDeviceSynchronize());
    prof.begin_mem2H();
    hipMemcpy(&aResult.count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    prof.end_mem2H();

    hipFree(d_label);
    hipFree(d_data);
    hipFree(d_count);
    return prof.retreive();
};

REGISTER_CLASS(IAccuracy,ReferenceAccuracy);