#include "cuda-utils.hpp"

void check_cuda_error(hipError_t error_code,const char* file, int line){
    if(error_code != hipSuccess){
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

void setup_gpu(){
    hipSetDevice(0);
}

void reset_state(){
    hipDeviceReset();
}

CudaProfiling::CudaProfiling(){
    CHECK_CUDA(hipEventCreate(&memstart2D));
    CHECK_CUDA(hipEventCreate(&memstop2D));
    CHECK_CUDA(hipEventCreate(&memstart2H));
    CHECK_CUDA(hipEventCreate(&memstop2H));
    CHECK_CUDA(hipEventCreate(&computestart));
    CHECK_CUDA(hipEventCreate(&computestop));
};

CudaProfiling::~CudaProfiling(){
    CHECK_CUDA(hipEventDestroy(memstart2D));
    CHECK_CUDA(hipEventDestroy(memstop2D));
    CHECK_CUDA(hipEventDestroy(memstart2H));
    CHECK_CUDA(hipEventDestroy(memstop2H));
    CHECK_CUDA(hipEventDestroy(computestart));
    CHECK_CUDA(hipEventDestroy(computestop));
};

void CudaProfiling::begin_mem2D(){
    CHECK_CUDA(hipEventRecord(memstart2D));
};

void CudaProfiling::end_mem2D(){
    CHECK_CUDA(hipEventRecord(memstop2D));
};

void CudaProfiling::begin_mem2H(){
    CHECK_CUDA(hipEventRecord(memstart2H));
};

void CudaProfiling::end_mem2H(){
    CHECK_CUDA(hipEventRecord(memstop2H));
};


void CudaProfiling::begin_compute(){
    CHECK_CUDA(hipEventRecord(computestart));
};

void CudaProfiling::end_compute(){
    CHECK_CUDA(hipEventRecord(computestop));
};


KernelStats CudaProfiling::retreive(){
    CHECK_CUDA(hipEventSynchronize(memstop2D));
    CHECK_CUDA(hipEventSynchronize(computestop));
    CHECK_CUDA(hipEventSynchronize(memstop2H));

    KernelStats stats;
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2D, memstart2D, memstop2D));
    CHECK_CUDA(hipEventElapsedTime(&stats.compute, computestart, computestop));
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2H, memstart2H, memstop2H));

    return stats;
}