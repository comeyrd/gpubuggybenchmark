#include "cuda-utils.hpp"
#include "gpu-utils.hpp"

void setup_gpu() {
    CHECK_CUDA(hipSetDevice(0));
}

void reset_gpu() {
    CHECK_CUDA(hipDeviceReset());
    // setup_gpu();
}

void check_cuda_error(hipError_t error_code, const char *file, int line) {
    if (error_code != hipSuccess) {
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

CudaProfiling::CudaProfiling(BaseSettings settings_) : settings(settings_) {
    CHECK_CUDA(hipEventCreate(&memstart2D));
    CHECK_CUDA(hipEventCreate(&memstop2D));
    CHECK_CUDA(hipEventCreate(&memstart2H));
    CHECK_CUDA(hipEventCreate(&memstop2H));
    warmupstart = new hipEvent_t[settings.warmup];
    warmupstop = new hipEvent_t[settings.warmup];
    repetitionstart = new hipEvent_t[settings.repetitions];
    repetitionstop = new hipEvent_t[settings.repetitions];
    for (int w = 0; w < settings.warmup; w++) {
        CHECK_CUDA(hipEventCreate(&warmupstart[w]));
        CHECK_CUDA(hipEventCreate(&warmupstop[w]));
    }
    for (int r = 0; r < settings.repetitions; r++) {
        CHECK_CUDA(hipEventCreate(&repetitionstart[r]));
        CHECK_CUDA(hipEventCreate(&repetitionstop[r]));
    }
};

CudaProfiling::~CudaProfiling() {
    if (!destroy) {
        try {
            CHECK_CUDA(hipEventDestroy(memstart2D));
            CHECK_CUDA(hipEventDestroy(memstop2D));
            CHECK_CUDA(hipEventDestroy(memstart2H));
            CHECK_CUDA(hipEventDestroy(memstop2H));
            for (int w = 0; w < settings.warmup; w++) {
                CHECK_CUDA(hipEventDestroy(warmupstart[w]));
                CHECK_CUDA(hipEventDestroy(warmupstop[w]));
            }
            for (int r = 0; r < settings.repetitions; r++) {
                CHECK_CUDA(hipEventDestroy(repetitionstart[r]));
                CHECK_CUDA(hipEventDestroy(repetitionstop[r]));
            }
        } catch (std::exception &e) {
            // std::cerr << "Error destroying Cuda profiler" << e.what()<<std::endl;
        }
    }
};

void CudaProfiling::begin_mem2D() {
    CHECK_CUDA(hipEventRecord(memstart2D));
};

void CudaProfiling::end_mem2D() {
    CHECK_CUDA(hipEventRecord(memstop2D));
};

void CudaProfiling::begin_mem2H() {
    CHECK_CUDA(hipEventRecord(memstart2H));
};

void CudaProfiling::end_mem2H() {
    CHECK_CUDA(hipEventRecord(memstop2H));
};

void CudaProfiling::begin_warmup() {
    CHECK_CUDA(hipEventRecord(warmupstart[nb_w]));
};
void CudaProfiling::end_warmup() {
    CHECK_CUDA(hipEventRecord(warmupstop[nb_w]));
    nb_w++;
};
void CudaProfiling::begin_repetition() {
    CHECK_CUDA(hipEventRecord(repetitionstart[nb_r]));
};
void CudaProfiling::end_repetition() {
    CHECK_CUDA(hipEventRecord(repetitionstop[nb_r]));
    nb_r++;
};

KernelStats CudaProfiling::retreive() {
    CHECK_CUDA(hipEventSynchronize(memstop2D));
    CHECK_CUDA(hipEventSynchronize(memstop2H));

    KernelStats stats(settings);
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2D, memstart2D, memstop2D));
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2H, memstart2H, memstop2H));
    for (int w = 0; w < nb_w; w++) {
        CHECK_CUDA(hipEventElapsedTime(&stats.warmup_duration[w], warmupstart[w], warmupstop[w]));
    }
    for (int r = 0; r < nb_r; r++) {
        CHECK_CUDA(hipEventElapsedTime(&stats.repetitions_duration[r], repetitionstart[r], repetitionstop[r]));
    }
    stats.nb_r = nb_r;
    stats.nb_w = nb_w;
    if (!destroy) {
        CHECK_CUDA(hipEventDestroy(memstart2D));
        CHECK_CUDA(hipEventDestroy(memstop2D));
        CHECK_CUDA(hipEventDestroy(memstart2H));
        CHECK_CUDA(hipEventDestroy(memstop2H));
        for (int w = 0; w < settings.warmup; w++) {
            CHECK_CUDA(hipEventDestroy(warmupstart[w]));
            CHECK_CUDA(hipEventDestroy(warmupstop[w]));
        }
        for (int r = 0; r < settings.repetitions; r++) {
            CHECK_CUDA(hipEventDestroy(repetitionstart[r]));
            CHECK_CUDA(hipEventDestroy(repetitionstop[r]));
        }
        destroy = true;
    }
    return stats;
}