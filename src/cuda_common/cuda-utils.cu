#include "cuda-utils.hpp"
#include "gpu-utils.hpp"

void setup_gpu(){
    CHECK_CUDA(hipSetDevice(0));
}

void reset_gpu(){
    CHECK_CUDA(hipDeviceReset());
}

void check_cuda_error(hipError_t error_code,const char* file, int line){
    if(error_code != hipSuccess){ 
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

CudaProfiling::CudaProfiling(){
    CHECK_CUDA(hipEventCreate(&memstart2D));
    CHECK_CUDA(hipEventCreate(&memstop2D));
    CHECK_CUDA(hipEventCreate(&memstart2H));
    CHECK_CUDA(hipEventCreate(&memstop2H));
    CHECK_CUDA(hipEventCreate(&computestart));
    CHECK_CUDA(hipEventCreate(&computestop));
};

CudaProfiling::~CudaProfiling(){
    if(!destroy){
        try{
            CHECK_CUDA(hipEventDestroy(memstart2D));
            CHECK_CUDA(hipEventDestroy(memstop2D));
            CHECK_CUDA(hipEventDestroy(memstart2H));
            CHECK_CUDA(hipEventDestroy(memstop2H));
            CHECK_CUDA(hipEventDestroy(computestart));
            CHECK_CUDA(hipEventDestroy(computestop));
        }catch(std::exception &e){
            std::cerr << "Error destroying Cuda profiler" << e.what()<<std::endl;
        }
    }
    
};

void CudaProfiling::begin_mem2D(){
    CHECK_CUDA(hipEventRecord(memstart2D));
};

void CudaProfiling::end_mem2D(){
    CHECK_CUDA(hipEventRecord(memstop2D));
};

void CudaProfiling::begin_mem2H(){
    CHECK_CUDA(hipEventRecord(memstart2H));
};

void CudaProfiling::end_mem2H(){
    CHECK_CUDA(hipEventRecord(memstop2H));
};


void CudaProfiling::begin_compute(){
    CHECK_CUDA(hipEventRecord(computestart));
};

void CudaProfiling::end_compute(){
    CHECK_CUDA(hipEventRecord(computestop));
};


KernelStats CudaProfiling::retreive(){
    CHECK_CUDA(hipEventSynchronize(memstop2D));
    CHECK_CUDA(hipEventSynchronize(computestop));
    CHECK_CUDA(hipEventSynchronize(memstop2H));

    KernelStats stats;
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2D, memstart2D, memstop2D));
    CHECK_CUDA(hipEventElapsedTime(&stats.compute, computestart, computestop));
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2H, memstart2H, memstop2H));
    if(!destroy){ 
        CHECK_CUDA(hipEventDestroy(memstart2D));
        CHECK_CUDA(hipEventDestroy(memstop2D));
        CHECK_CUDA(hipEventDestroy(memstart2H));
        CHECK_CUDA(hipEventDestroy(memstop2H));
        CHECK_CUDA(hipEventDestroy(computestart));
        CHECK_CUDA(hipEventDestroy(computestop));
        destroy = true;
    }
   
    return stats;
}