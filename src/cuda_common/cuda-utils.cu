#include "cuda-utils.hpp"
#include <cuda/std/chrono>

event_t **allocate_event_array(int count) {
    event_t **events = new event_t *[count];
    for (int i = 0; i < count; i++) {
        events[i] = new event_t;
        CHECK_CUDA(hipEventCreate(&events[i]->native));
    }
    return events;
}

void free_event_array(event_t **events, int count) {
    for (int i = 0; i < count; i++) {
        CHECK_CUDA(hipEventDestroy(events[i]->native));
        delete events[i];
    }
    delete[] events;
}

GpuStream::GpuStream() {
    stream = new stream_t;
    CHECK_CUDA(hipStreamCreate(&stream->native));
}
GpuStream::~GpuStream() {
    CHECK_CUDA(hipStreamDestroy(stream->native));
    delete stream;
}
bool GpuStream::get_stream_availability(){
    hipError_t stream_status = hipStreamQuery(stream->native);
    if(stream_status == hipSuccess){
          return true;
    }else{
        std::cout << hipGetErrorString(stream_status) << std::endl;
        return false;
    }
}
void GpuStream::synchronize(){
    CHECK_CUDA(hipStreamSynchronize(stream->native));
}

void setup_gpu() {
    CHECK_CUDA(hipSetDevice(0));
}

void reset_gpu() {
    CHECK_CUDA(hipDeviceReset());
    // setup_gpu();
}

void check_cuda_error(hipError_t error_code, const char *file, int line) {
    if (error_code != hipSuccess) {
        std::string msg = std::string("CUDA Error : ") + hipGetErrorString(error_code) + std::string(" in : ") + file + std::string(" line ") + std::to_string(line);
        throw std::runtime_error(msg);
    }
}

GpuEventTimer::GpuEventTimer(int warmup, int repetitions, stream_t *gpustream) : m_warmups(warmup), m_repetitions(repetitions), stream(gpustream) {
    memstart2D = new event_t;
    memstop2D = new event_t;
    memstart2H = new event_t;
    memstop2H = new event_t;
    CHECK_CUDA(hipEventCreate(&memstart2D->native));
    CHECK_CUDA(hipEventCreate(&memstop2D->native));
    CHECK_CUDA(hipEventCreate(&memstart2H->native));
    CHECK_CUDA(hipEventCreate(&memstop2H->native));
    warmupstart = allocate_event_array(m_warmups);
    warmupstop = allocate_event_array(m_warmups);
    repetitionstart = allocate_event_array(m_repetitions);
    repetitionstop = allocate_event_array(m_repetitions);
};

GpuEventTimer::~GpuEventTimer() {
    try {
        CHECK_CUDA(hipEventDestroy(memstart2D->native));
        CHECK_CUDA(hipEventDestroy(memstop2D->native));
        CHECK_CUDA(hipEventDestroy(memstart2H->native));
        CHECK_CUDA(hipEventDestroy(memstop2H->native));
        free_event_array(warmupstart, m_warmups);
        free_event_array(warmupstop, m_warmups);
        free_event_array(repetitionstart, m_repetitions);
        free_event_array(repetitionstop, m_repetitions);
    } catch (std::exception &e) {
        std::cerr << "Error destroying Cuda profiler" << e.what() << std::endl;
    }
};

void GpuEventTimer::begin_mem2D() {
    CHECK_CUDA(hipEventRecord(memstart2D->native, stream->native));
};

void GpuEventTimer::end_mem2D() {
    CHECK_CUDA(hipEventRecord(memstop2D->native, stream->native));
};

void GpuEventTimer::begin_mem2H() {
    CHECK_CUDA(hipEventRecord(memstart2H->native, stream->native));
};

void GpuEventTimer::end_mem2H() {
    CHECK_CUDA(hipEventRecord(memstop2H->native, stream->native));
};

void GpuEventTimer::begin_warmup() {
    CHECK_CUDA(hipEventRecord(warmupstart[nb_w]->native, stream->native));
};
void GpuEventTimer::end_warmup() {
    CHECK_CUDA(hipEventRecord(warmupstop[nb_w]->native, stream->native));
    nb_w++;
};
void GpuEventTimer::begin_repetition() {
    CHECK_CUDA(hipEventRecord(repetitionstart[nb_r]->native, stream->native));
};
void GpuEventTimer::end_repetition() {
    CHECK_CUDA(hipEventRecord(repetitionstop[nb_r]->native, stream->native));
    nb_r++;
};

KernelStats GpuEventTimer::retreive() {
    CHECK_CUDA(hipEventSynchronize(memstop2D->native));
    CHECK_CUDA(hipEventSynchronize(memstart2D->native));
    CHECK_CUDA(hipEventSynchronize(memstart2H->native));
    CHECK_CUDA(hipEventSynchronize(memstop2H->native));
    KernelStats stats(nb_w,nb_r);
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2D, memstart2D->native, memstop2D->native));
    CHECK_CUDA(hipEventElapsedTime(&stats.memcpy2H, memstart2H->native, memstop2H->native));
    for (int w = 0; w < nb_w; w++) {
        CHECK_CUDA(hipEventElapsedTime(&stats.warmup_duration[w], warmupstart[w]->native, warmupstop[w]->native));
    }
    for (int r = 0; r < nb_r; r++) {
        CHECK_CUDA(hipEventElapsedTime(&stats.repetitions_duration[r], repetitionstart[r]->native, repetitionstop[r]->native));
    }
    stats.nb_r = nb_r;
    stats.nb_w = nb_w;
    return stats;
}

l2flushr::l2flushr() : cs() {
    int dev_id{};
    CHECK_CUDA(hipGetDevice(&dev_id));
    CHECK_CUDA(hipDeviceGetAttribute(&buffer_size, hipDeviceAttributeL2CacheSize, dev_id));
    if (buffer_size > 0) {
        void *buffer = l2_buffer;
        CHECK_CUDA(hipMalloc(&buffer, static_cast<std::size_t>(buffer_size)));
        l2_buffer = reinterpret_cast<int *>(buffer);
    }
}
l2flushr::~l2flushr() {
    if (l2_buffer) {
        CHECK_CUDA(hipFree(l2_buffer));
    }
}
void l2flushr::flush(stream_t *stream) {
    CHECK_CUDA(hipMemsetAsync(l2_buffer, 0, static_cast<std::size_t>(buffer_size), stream->native));
}